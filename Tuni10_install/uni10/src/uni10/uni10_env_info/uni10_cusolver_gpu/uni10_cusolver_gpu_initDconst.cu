#include "uni10/uni10_sys_info/uni10_cusolver_gpu/uni10_cusolver_gpu_Dconst.cuh"
#include "uni10/uni10_env_info/uni10_cusolver_gpu/uni10_cusolver_gpu_initDconst.h"

namespace uni10{

  void uni10_init_device_const(sysinfo_gpu& uni10_sys_info){

    uni10_int memsize = sizeof(uni10_int);
    hipMemcpyToSymbol(HIP_SYMBOL( MAXGRIDSIZE_X), &uni10_sys_info.maxGridSize_x, memsize);
    hipMemcpyToSymbol(HIP_SYMBOL( MAXGRIDSIZE_Y), &uni10_sys_info.maxGridSize_y, memsize);
    hipMemcpyToSymbol(HIP_SYMBOL( MAXGRIDSIZE_Z), &uni10_sys_info.maxGridSize_z, memsize);
    hipMemcpyToSymbol(HIP_SYMBOL( MAXTHREADSPERBLOCK), &uni10_sys_info.maxThreadsPerBlock, memsize);
    hipMemcpyToSymbol(HIP_SYMBOL( MAXTHREADSDIM_X), &uni10_sys_info.maxThreadsDim_x, memsize);
    hipMemcpyToSymbol(HIP_SYMBOL( MAXTHREADSDIM_Y), &uni10_sys_info.maxThreadsDim_y, memsize);
    hipMemcpyToSymbol(HIP_SYMBOL( MAXTHREADSDIM_Z), &uni10_sys_info.maxThreadsDim_z, memsize);

  }

  std::map<std::string, uni10_int> uni10_get_device_const(){
    
    uni10_int memsize = sizeof(uni10_int);

    std::map<std::string, uni10_int> dev_info;
    uni10_int para;
    hipMemcpyFromSymbol( &para, HIP_SYMBOL(MAXGRIDSIZE_X), memsize);
    dev_info["MAXGRIDSIZE_X"] = para;
    hipMemcpyFromSymbol( &para, HIP_SYMBOL(MAXGRIDSIZE_Y), memsize);
    dev_info["MAXGRIDSIZE_Y"] = para;
    hipMemcpyFromSymbol( &para, HIP_SYMBOL(MAXGRIDSIZE_Z), memsize);
    dev_info["MAXGRIDSIZE_Z"] = para;
    hipMemcpyFromSymbol( &para, HIP_SYMBOL(MAXTHREADSPERBLOCK), memsize);
    dev_info["MAXTHREADSPERBLOCK"] = para;
    hipMemcpyFromSymbol( &para, HIP_SYMBOL(MAXTHREADSDIM_X), memsize);
    dev_info["MAXTHREADSDIM_X"] = para;
    hipMemcpyFromSymbol( &para, HIP_SYMBOL(MAXTHREADSDIM_Y), memsize);
    dev_info["MAXTHREADSDIM_Y"] = para;
    hipMemcpyFromSymbol( &para, HIP_SYMBOL(MAXTHREADSDIM_Z), memsize);
    dev_info["MAXTHREADSDIM_Z"] = para;

    return dev_info;

  }

}; // End of uni10 namespace

